#include "hip/hip_runtime.h"
#include "filter.h"

template <typename T>
__global__ void GaussianFilterSTY_GPU_kernel(T* target, T* source, int width, int height, T scale, T d, BoundaryCondition boundary, bool add) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    if (x == 0 && y == 0) printf("GPU kernel %f\n", source[0]);
}

template <typename T>
void GaussianFilterSTY_GPU(T* target, T* source, int width, int height, T scale, T d, BoundaryCondition boundary, bool add) {
    dim3 block_size;
	block_size.x = 16;
	block_size.y = 16;
	block_size.z = 1;

	dim3 grid_size;
	grid_size.x = (width + block_size.x - 1) / block_size.x;
	grid_size.y = (height + block_size.y - 1) / block_size.y;
    grid_size.z = 1;
    
	GaussianFilterSTY_GPU_kernel<<<grid_size, block_size>>>((T *)target, (T *)source, width, height, (T)scale, (T)d, boundary, add);
	CHECK_LAUNCH_ERROR();
}

template <typename T>
void GaussianFilterSTX_GPU(T* target, T* source, int width, int height, T scale, T d, BoundaryCondition boundary, bool add) {

}

template void GaussianFilterSTY_GPU<float>(float* target, float* source, int width, int height, float scale, float d, BoundaryCondition boundary, bool add);
template void GaussianFilterSTY_GPU<double>(double* target, double* source, int width, int height, double scale, double d, BoundaryCondition boundary, bool add);

template void GaussianFilterSTX_GPU<float>(float* target, float* source, int width, int height, float scale, float d, BoundaryCondition boundary, bool add);
template void GaussianFilterSTX_GPU<double>(double* target, double* source, int width, int height, double scale, double d, BoundaryCondition boundary, bool add);
