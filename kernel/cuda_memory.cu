﻿#include "cuda_memory.h"
#include "kernel_cu.h"

void freeDeviceData(hipArray_t &d_ptr)
{
	if (d_ptr == (hipArray_t)NULL) return;
	allocationLogger.free(d_ptr);
	{
		checkCudaErrors(hipFreeArray(d_ptr));
	}
	d_ptr = (hipArray_t)NULL;
}
